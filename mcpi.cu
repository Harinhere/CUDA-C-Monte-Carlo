//Author: Harindranath Ambalampitiya, PhD(Theoretical atomic and molecular physics)
//

#include <hip/hip_runtime.h>
#include <iostream>
#include<math.h>
#include<stdio.h>
#include<ctime>
#include<cstdlib>
#include <chrono> 
#include<hiprand/hiprand_kernel.h>

using namespace std;
using namespace std::chrono;

// initialize random_number generator on the device
//each thread gets the same seed,but different sequence
__global__ void rng_init(hiprandState *state,int seed,int n)
{
	int id=blockIdx.x*blockDim.x+threadIdx.x;
	if(id<n)
	{
		hiprand_init(seed, id, 0, &state[id]);
	}
} 

//Let's calculate pi on the device
__global__ void mcpiKernel(hiprandState *state,int *a, int NSW,float r)
{		
	//Monte carlo region
	float xmin=-r,xmax=r,ymin=-r,ymax=r;
	int idx=blockIdx.x*blockDim.x+threadIdx.x;
	//copy state to local memory
	hiprandState localState = state[idx];
	
    int sum_in=0;
	for(int i=1;i<=NSW;i++)
	{
		//generate random numbers in the uniform grid (0,1]
		//for both x and y coordinates
		float ran0 = hiprand_uniform(&localState);
		float ran1 = hiprand_uniform(&localState);
		float x=xmin+(xmax-xmin)*ran0;
		float y=ymin+(ymax-ymin)*ran1;
		float d=sqrt(x*x+y*y);
		//printf("x,y: %f \t %f \n",x,y);
		if(d<=r)
			sum_in=sum_in+1;
	}
	//copy local memory to global
	state[idx] = localState;
	a[idx]=sum_in;
	//printf("inside: %i \n",sum_in);
}



float cudaPi(int N)
{
	// number of threads and blocks
	int block_size=256;
	int n_blocks=128;
	int n_procs=n_blocks*block_size;
	//memory allocation in the host and device
	
	size_t size=n_procs *sizeof(int);
	int* a_h=(int*)malloc(size);
	int* a_d;
	hipMalloc((void **) &a_d, size);
	//random_states
	hiprandState *devStates;
	hipMalloc((void **) &devStates, n_procs *sizeof(hiprandState));
	
	
	//number of sweeps that each graphic processor gets
	int nsw=N/n_procs+(N%n_procs==0 ? 0:1);
	
	//initialize the random numbers
	int s=12345;//seed
	rng_init<<<n_blocks,block_size>>>(devStates, s, n_procs);
	
	//pass it to parallel processing
	//each parallel unit counts how many points lie inside the circle
	float r=0.5;//circle radius
	
	mcpiKernel<<<n_blocks,block_size>>>(devStates,a_d,nsw,r);
	
	hipMemcpy(a_h,a_d, sizeof(int)*n_procs,hipMemcpyDeviceToHost);
	
	//number of points inside/outsie the circle
	float sum_in=0.;
	float sum_out=nsw*n_procs;
	
	for(int i=0;i<n_procs;i++)sum_in +=(float)a_h[i];
	
	//printf("sum in is %f \n sum out is %f \n",sum_in,sum_out);
	
	float pii=4.0f*(sum_in/sum_out);
	//now free-up the space
	free(a_h);
	hipFree(a_d);
	
	return pii;
}

int main()
{
	auto start = high_resolution_clock::now();
	int N=10000000;
	float pii=cudaPi(N);
	auto stop = high_resolution_clock::now(); 
	auto duration = duration_cast<milliseconds>(stop - start);	
	printf("Pi value is: %f \n ",pii);	
	cout<<"Duration (ms)"<<"\t"<<duration.count()<<endl;
}